#include "hip/hip_runtime.h"
// dipole_field.cu

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <math.h>
#include "dipole_field.hpp"

__device__ vec3 device_vec3_sub(vec3 a, vec3 b) {
    return {a.x - b.x, a.y - b.y, a.z - b.z};
}

__device__ double device_vec3_norm(vec3 v) {
    return sqrt(v.x * v.x + v.y * v.y + v.z * v.z);
}

__device__ vec3 device_vec3_normalize(vec3 v) {
    double norm = device_vec3_norm(v);
    return {v.x / norm, v.y / norm, v.z / norm};
}

__device__ hipDoubleComplex dot_cvec3(vec3 a, cvec3 b) {
    return hipCadd(hipCadd(
        hipCmul(make_hipDoubleComplex(a.x, 0.0), b.x),
        hipCmul(make_hipDoubleComplex(a.y, 0.0), b.y)),
        hipCmul(make_hipDoubleComplex(a.z, 0.0), b.z));
}

__device__ cvec3 cross_cvec3(cvec3 a, cvec3 b) {
    return {
        hipCsub(hipCmul(a.y, b.z), hipCmul(a.z, b.y)),
        hipCsub(hipCmul(a.z, b.x), hipCmul(a.x, b.z)),
        hipCsub(hipCmul(a.x, b.y), hipCmul(a.y, b.x))
    };
}

__device__ cvec3 scale_cvec3(cvec3 v, hipDoubleComplex s) {
    return {
        hipCmul(v.x, s),
        hipCmul(v.y, s),
        hipCmul(v.z, s)
    };
}

__device__ cvec3 add_cvec3(cvec3 a, cvec3 b) {
    return {
        hipCadd(a.x, b.x),
        hipCadd(a.y, b.y),
        hipCadd(a.z, b.z)
    };
}

__device__ cvec3 cross_vec3_cvec3(vec3 a, cvec3 b) {
    return {
        hipCsub(hipCmul(make_hipDoubleComplex(a.y, 0), b.z), hipCmul(make_hipDoubleComplex(a.z, 0), b.y)),
        hipCsub(hipCmul(make_hipDoubleComplex(a.z, 0), b.x), hipCmul(make_hipDoubleComplex(a.x, 0), b.z)),
        hipCsub(hipCmul(make_hipDoubleComplex(a.x, 0), b.y), hipCmul(make_hipDoubleComplex(a.y, 0), b.x))
    };
}

__device__ void print_vec3(const char* name, vec3 v) {
    printf("%s: (%e, %e, %e)\n", name, v.x, v.y, v.z);
}

__device__ void print_cvec3(const char* name, cvec3 v) {
    printf("%s: (%e + %ei, %e + %ei, %e + %ei)\n", 
           name,
           hipCreal(v.x), hipCimag(v.x),
           hipCreal(v.y), hipCimag(v.y),
           hipCreal(v.z), hipCimag(v.z));
}

__device__ void print_complex(const char* name, hipDoubleComplex z) {
    printf("%s: %e + %ei\n", name, hipCreal(z), hipCimag(z));
}

__global__ void compute_field(
    const vec3* dipole_pos, const cvec3* dipole_mom, int N_dip,
    const vec3* obs_pos, cvec3* E_out, cvec3* B_out, int N_obs,
    double k, double prefac)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N_obs) return;

    vec3 r_obs = obs_pos[i];
    cvec3 E = {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0)};
    cvec3 B = {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0)};

    double c = 299792458.0;
    double omega = k * c;
    
    // Only print debug info if this is our special point at (0,0,1e-6)
    bool is_debug_point = false; //(fabs(r_obs.x) < 1e-10 && fabs(r_obs.y) < 1e-10 && fabs(r_obs.z - 1e-6) < 1e-10);
    
    if (is_debug_point) {
        printf("\n=== Computing fields at observation point ===\n");
        print_vec3("Observation point r_obs", r_obs);
        printf("Wave number k: %e\n", k);
        printf("Prefactor: %e\n", prefac);
    }

    for (int j = 0; j < N_dip; ++j) {
        vec3 r_dip = dipole_pos[j];
        cvec3 p = dipole_mom[j];
        
        if (is_debug_point) {
            printf("\n--- Dipole %d contribution ---\n", j);
            print_vec3("Dipole position r_dip", r_dip);
            print_cvec3("Dipole moment p", p);
        }

        vec3 r = device_vec3_sub(r_obs, r_dip);
        double R = device_vec3_norm(r);
        vec3 n = device_vec3_normalize(r);
        
        if (is_debug_point) {
            print_vec3("Distance vector r", r);
            printf("Distance magnitude R: %e\n", R);
            print_vec3("Unit vector n", n);
        }

        // Electric field computation
        cvec3 n_cross_p = cross_vec3_cvec3(n, p);
        if (is_debug_point) print_cvec3("n × p", n_cross_p);
        
        cvec3 term1 = cross_vec3_cvec3(n, n_cross_p);
        term1 = scale_cvec3(term1, make_hipDoubleComplex(-k * k / R, 0.0));
        if (is_debug_point) print_cvec3("Far-field term (k²(n × p) × n)/R", term1);

        hipDoubleComplex n_dot_p = dot_cvec3(n, p);
        if (is_debug_point) print_complex("n · p", n_dot_p);
        
        cvec3 three_n_n_dot_p = {
            hipCmul(make_hipDoubleComplex(3 * n.x, 0.0), n_dot_p),
            hipCmul(make_hipDoubleComplex(3 * n.y, 0.0), n_dot_p),
            hipCmul(make_hipDoubleComplex(3 * n.z, 0.0), n_dot_p),
        };
        if (is_debug_point) print_cvec3("3n(n · p)", three_n_n_dot_p);
        
        cvec3 term2_vec = {
            hipCsub(three_n_n_dot_p.x, p.x),
            hipCsub(three_n_n_dot_p.y, p.y),
            hipCsub(three_n_n_dot_p.z, p.z),
        };
        if (is_debug_point) print_cvec3("3n(n · p) - p", term2_vec);

        hipDoubleComplex scalar = hipCdiv(
            hipCsub(make_hipDoubleComplex(1.0, 0.0), make_hipDoubleComplex(0.0, k * R)),
            make_hipDoubleComplex(R * R * R, 0.0)
        );
        if (is_debug_point) print_complex("Near-field scalar (1-ikR)/R³", scalar);
        
        cvec3 term2 = scale_cvec3(term2_vec, scalar);
        if (is_debug_point) print_cvec3("Near-field term [(3n(n · p) - p)(1-ikR)/R³]", term2);

        hipDoubleComplex phase = make_hipDoubleComplex(cos(k * R), sin(k * R));
        if (is_debug_point) print_complex("Phase factor e^(ikR)", phase);
        
        cvec3 total_E = add_cvec3(term1, term2);
        total_E = scale_cvec3(total_E, hipCmul(phase, make_hipDoubleComplex(prefac, 0.0)));
        if (is_debug_point) print_cvec3("Total E field contribution from this dipole", total_E);

        // Magnetic field computation
        double mu0 = 1.25663706212e-6;
        hipDoubleComplex factor = hipCmul(
            make_hipDoubleComplex(0.0, mu0 * omega),
            hipCsub(make_hipDoubleComplex(1.0 / R, 0.0), make_hipDoubleComplex(0.0, k))
        );
        if (is_debug_point) print_complex("B-field factor -iμ₀ω(1/R - ik)", factor);

        hipDoubleComplex scale = hipCdiv(factor, make_hipDoubleComplex(4 * M_PI * R, 0.0));
        scale = hipCmul(scale, phase);
        if (is_debug_point) print_complex("B-field scaling factor", scale);

        cvec3 B_contrib = scale_cvec3(cross_vec3_cvec3(n, p), scale);
        if (is_debug_point) print_cvec3("B field contribution from this dipole", B_contrib);

        E = add_cvec3(E, total_E);
        B = add_cvec3(B, B_contrib);
    }

    if (is_debug_point) {
        printf("\n=== Final Results ===\n");
        print_cvec3("Total E field", E);
        print_cvec3("Total B field", B);
    }
    
    E_out[i] = E;
    B_out[i] = B;
}