#include "hip/hip_runtime.h"
// sample_fields.cu
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include "dipole_field.hpp"
#include "load_dipole_data.hpp"

__device__ hipDoubleComplex complex_conj(hipDoubleComplex z) {
    return make_hipDoubleComplex(hipCreal(z), -hipCimag(z));
}

__device__ double real_cross_dot(const cvec3& E, const cvec3& B) {
    // Real part of (E x B*) . ẑ
    hipDoubleComplex ExBy = hipCmul(E.x, complex_conj(B.y));
    hipDoubleComplex EyBx = hipCmul(E.y, complex_conj(B.x));
    hipDoubleComplex ExBz = hipCmul(E.x, complex_conj(B.z));
    hipDoubleComplex EzBx = hipCmul(E.z, complex_conj(B.x));
    hipDoubleComplex EyBz = hipCmul(E.y, complex_conj(B.z));
    hipDoubleComplex EzBy = hipCmul(E.z, complex_conj(B.y));

    hipDoubleComplex S_z = hipCsub(ExBy, EyBx); // (E x B*) . z
    return hipCreal(S_z);
}

__global__ void compute_poynting_flux(const cvec3* E, const cvec3* B, double* S_out, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;
    double mu0 = 1.25663706212e-6;
    S_out[i] = 0.5 * real_cross_dot(E[i], B[i]) / mu0;
}

int main(int argc, char** argv) {
    if (argc < 3) {
        std::cerr << "Usage: " << argv[0] << " dipole_data.csv frequency_in_Hz" << std::endl;
        return 1;
    }

    // Load dipole data from CSV
    std::vector<vec3> host_positions;
    std::vector<cvec3> host_dipoles;
    load_dipole_data(argv[1], host_positions, host_dipoles);

    int N_dip = host_positions.size();

    // Define sampling grid (example: 300x300 grid at z = 1000nm)
    const int Nx = 100, Ny = 100;
    const double z_sample = 5000e-9;
    const double center_x = 5e-6;
    const double center_y = 5e-6;
    const double grid_size = 9e-6; // 10 micron patch
    const double dx = grid_size / (Nx - 1);
    const double dy = grid_size / (Ny - 1);

    std::vector<vec3> host_obs(Nx * Ny);
    for (int ix = 0; ix < Nx; ++ix) {
        for (int iy = 0; iy < Ny; ++iy) {
            double x = center_x + dx * (ix - Nx/2);
            double y = center_y + dy * (iy - Ny/2);
            host_obs[iy * Nx + ix] = {x, y, z_sample};

            // std::cout << x << " " << y << " " << z_sample << std::endl;
        }
    }

    int N_obs = Nx * Ny;
    std::vector<cvec3> host_E(N_obs);
    std::vector<cvec3> host_B(N_obs);
    std::vector<double> host_S(N_obs);

    // Allocate and copy data to device
    vec3* d_positions;
    cvec3* d_dipoles;
    vec3* d_obs;
    cvec3* d_E;
    cvec3* d_B;
    double* d_S;
    hipMalloc(&d_positions, N_dip * sizeof(vec3));
    hipMalloc(&d_dipoles, N_dip * sizeof(cvec3));
    hipMalloc(&d_obs, N_obs * sizeof(vec3));
    hipMalloc(&d_E, N_obs * sizeof(cvec3));
    hipMalloc(&d_B, N_obs * sizeof(cvec3));
    hipMalloc(&d_S, N_obs * sizeof(double));

    hipMemcpy(d_positions, host_positions.data(), N_dip * sizeof(vec3), hipMemcpyHostToDevice);
    hipMemcpy(d_dipoles, host_dipoles.data(), N_dip * sizeof(cvec3), hipMemcpyHostToDevice);
    hipMemcpy(d_obs, host_obs.data(), N_obs * sizeof(vec3), hipMemcpyHostToDevice);

    // Set physical parameters
    // Read frequency from command line
    double frequency = std::atof(argv[2]);
    double c = 299792458.0; // speed of light in vacuum
    double lambda = c / frequency;
    double k = 2 * M_PI / lambda;
    double prefac = 1.0 / (4 * M_PI * 8.854187817e-12); // 1/(4pi*epsilon0)
    double Z0 = 376.73; // Impedance of free space

    // Launch field kernel
    dim3 blockSize(256);
    dim3 gridSize((N_obs + blockSize.x - 1) / blockSize.x);
    compute_field<<<gridSize, blockSize>>>(d_positions, d_dipoles, N_dip, d_obs, d_E, d_B, N_obs, k, prefac);
    hipDeviceSynchronize();

    // Launch Poynting kernel
    compute_poynting_flux<<<gridSize, blockSize>>>(d_E, d_B, d_S, N_obs);
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(host_E.data(), d_E, N_obs * sizeof(cvec3), hipMemcpyDeviceToHost);
    hipMemcpy(host_B.data(), d_B, N_obs * sizeof(cvec3), hipMemcpyDeviceToHost);
    hipMemcpy(host_S.data(), d_S, N_obs * sizeof(double), hipMemcpyDeviceToHost);

    // Integrate Poynting flux
    double total_flux = 0.0;
    for (int i = 0; i < N_obs; ++i) {
        total_flux += host_S[i];
        // std::cout << host_S[i] << std::endl;
    }
    double avg_flux = total_flux / N_obs;
    total_flux *= dx * dy;

    std::cout << "(" << frequency << "," << avg_flux*2.0*Z0 << ")," << std::endl;

    // Add single point calculation
    // vec3 single_point = {0.0, 0.0, 1e-6};  // Point of interest
    // std::vector<vec3> single_obs(1, single_point);
    // std::vector<cvec3> single_E(1);
    // std::vector<cvec3> single_B(1);

    // // Allocate device memory for single point
    // vec3* d_single_obs;
    // cvec3* d_single_E;
    // cvec3* d_single_B;
    // hipMalloc(&d_single_obs, sizeof(vec3));
    // hipMalloc(&d_single_E, sizeof(cvec3));
    // hipMalloc(&d_single_B, sizeof(cvec3));

    // // Copy single observation point to device
    // hipMemcpy(d_single_obs, single_obs.data(), sizeof(vec3), hipMemcpyHostToDevice);

    // // Compute fields at single point
    // compute_field<<<1, 1>>>(d_positions, d_dipoles, N_dip, d_single_obs, d_single_E, d_single_B, 1, k, prefac);
    // hipDeviceSynchronize();

    // // Copy results back
    // hipMemcpy(single_E.data(), d_single_E, sizeof(cvec3), hipMemcpyDeviceToHost);
    // hipMemcpy(single_B.data(), d_single_B, sizeof(cvec3), hipMemcpyDeviceToHost);

    // // Print single point results
    // std::cout << "\nFields at point (0, 0, 1e-6):" << std::endl;
    // std::cout << "E-field (V/m): (" 
    //           << hipCreal(single_E[0].x) << " + " << hipCimag(single_E[0].x) << "i, "
    //           << hipCreal(single_E[0].y) << " + " << hipCimag(single_E[0].y) << "i, "
    //           << hipCreal(single_E[0].z) << " + " << hipCimag(single_E[0].z) << "i)" << std::endl;
    // std::cout << "B-field (T): (" 
    //           << hipCreal(single_B[0].x) << " + " << hipCimag(single_B[0].x) << "i, "
    //           << hipCreal(single_B[0].y) << " + " << hipCimag(single_B[0].y) << "i, "
    //           << hipCreal(single_B[0].z) << " + " << hipCimag(single_B[0].z) << "i)" << std::endl;

    // // Free additional device memory
    // hipFree(d_single_obs);
    // hipFree(d_single_E);
    // hipFree(d_single_B);

    // Free device memory
    hipFree(d_positions);
    hipFree(d_dipoles);
    hipFree(d_obs);
    hipFree(d_E);
    hipFree(d_B);
    hipFree(d_S);

    return 0;
}
