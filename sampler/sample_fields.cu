#include "hip/hip_runtime.h"
// sample_fields.cu
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include "dipole_field.hpp"
#include "load_dipole_data.hpp"

__device__ hipDoubleComplex complex_conj(hipDoubleComplex z) {
    return make_hipDoubleComplex(hipCreal(z), -hipCimag(z));
}

__device__ double real_cross_dot(const cvec3& E, const cvec3& B) {
    // Real part of (E x B*) . ẑ
    hipDoubleComplex ExBy = hipCmul(E.x, complex_conj(B.y));
    hipDoubleComplex EyBx = hipCmul(E.y, complex_conj(B.x));
    hipDoubleComplex ExBz = hipCmul(E.x, complex_conj(B.z));
    hipDoubleComplex EzBx = hipCmul(E.z, complex_conj(B.x));
    hipDoubleComplex EyBz = hipCmul(E.y, complex_conj(B.z));
    hipDoubleComplex EzBy = hipCmul(E.z, complex_conj(B.y));

    hipDoubleComplex S_z = hipCsub(ExBy, EyBx); // (E x B*) . z
    return hipCreal(S_z);
}

__global__ void compute_poynting_flux(const cvec3* E, const cvec3* B, double* S_out, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;
    double mu0 = 1.25663706212e-6;
    S_out[i] = 0.5 * real_cross_dot(E[i], B[i]) / mu0;
}

int main(int argc, char** argv) {
    if (argc < 3) {
        std::cerr << "Usage: " << argv[0] << " dipole_data.csv frequency_in_Hz" << std::endl;
        return 1;
    }

    // Load dipole data from CSV
    std::vector<vec3> host_positions;
    std::vector<cvec3> host_dipoles;
    load_dipole_data(argv[1], host_positions, host_dipoles);

    int N_dip = host_positions.size();

    // Define sampling grid (example: 100x100 grid at z = -100nm)
    const int Nx = 300, Ny = 300;
    const double z_sample = 6000e-9;
    const double grid_size = 30e-6; // 30 micron patch
    const double dx = grid_size / (Nx - 1);
    const double dy = grid_size / (Ny - 1);

    std::vector<vec3> host_obs(Nx * Ny);
    for (int ix = 0; ix < Nx; ++ix) {
        for (int iy = 0; iy < Ny; ++iy) {
            double x = -grid_size/2 + dx * ix;
            double y = -grid_size/2 + dy * iy;
            host_obs[iy * Nx + ix] = {x, y, z_sample};
        }
    }

    int N_obs = Nx * Ny;
    std::vector<cvec3> host_E(N_obs);
    std::vector<cvec3> host_B(N_obs);
    std::vector<double> host_S(N_obs);

    // Allocate and copy data to device
    vec3* d_positions;
    cvec3* d_dipoles;
    vec3* d_obs;
    cvec3* d_E;
    cvec3* d_B;
    double* d_S;
    hipMalloc(&d_positions, N_dip * sizeof(vec3));
    hipMalloc(&d_dipoles, N_dip * sizeof(cvec3));
    hipMalloc(&d_obs, N_obs * sizeof(vec3));
    hipMalloc(&d_E, N_obs * sizeof(cvec3));
    hipMalloc(&d_B, N_obs * sizeof(cvec3));
    hipMalloc(&d_S, N_obs * sizeof(double));

    hipMemcpy(d_positions, host_positions.data(), N_dip * sizeof(vec3), hipMemcpyHostToDevice);
    hipMemcpy(d_dipoles, host_dipoles.data(), N_dip * sizeof(cvec3), hipMemcpyHostToDevice);
    hipMemcpy(d_obs, host_obs.data(), N_obs * sizeof(vec3), hipMemcpyHostToDevice);

    // Set physical parameters
    // Read frequency from command line
    double frequency = std::atof(argv[2]);
    double c = 299792458.0; // speed of light in vacuum
    double lambda = c / frequency;
    double k = 2 * M_PI / lambda;
    double prefac = 1.0 / (4 * M_PI * 8.854187817e-12); // 1/(4pi*epsilon0)

    // Launch field kernel
    dim3 blockSize(256);
    dim3 gridSize((N_obs + blockSize.x - 1) / blockSize.x);
    compute_field<<<gridSize, blockSize>>>(d_positions, d_dipoles, N_dip, d_obs, d_E, d_B, N_obs, k, prefac);
    hipDeviceSynchronize();

    // Launch Poynting kernel
    compute_poynting_flux<<<gridSize, blockSize>>>(d_E, d_B, d_S, N_obs);
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(host_E.data(), d_E, N_obs * sizeof(cvec3), hipMemcpyDeviceToHost);
    hipMemcpy(host_B.data(), d_B, N_obs * sizeof(cvec3), hipMemcpyDeviceToHost);
    hipMemcpy(host_S.data(), d_S, N_obs * sizeof(double), hipMemcpyDeviceToHost);

    // Integrate Poynting flux
    double total_flux = 0.0;
    for (int i = 0; i < N_obs; ++i) {
        total_flux += host_S[i];
    }
    total_flux *= dx * dy;

    std::cout << "(" << frequency << "," << total_flux << ")," << std::endl;
    // std::cout << "Total power transmitted through plane: " << total_flux << " W" << std::endl;
    // std::cout << "Total radiated power estimate: " << total_flux*2 << " W" << std::endl;

    // Free device memory
    hipFree(d_positions);
    hipFree(d_dipoles);
    hipFree(d_obs);
    hipFree(d_E);
    hipFree(d_B);
    hipFree(d_S);

    return 0;
}
